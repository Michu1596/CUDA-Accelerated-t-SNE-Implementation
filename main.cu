#include "hip/hip_runtime.h"
/* 
żeby intlisense od vsighta działało potrzben sa te pliki co sa w .vscode
*/
// CUDA runtime
#include <hip/hip_runtime.h>

// includes
#include <hip/hip_runtime_api.h>  // helper functions for CUDA error checking and initialization
#include <helper_functions.h>  // helper for shared functions common to CUDA Samples

#include <hip/hip_runtime.h>

#include <cassert>
#include <iostream>
#include <memory>
#include <random>

#include "helper_functions.h"
#include "consts.h"

double data[N * DIMENSIONS];
#define TRIANGLE(X, Y) ( X < Y ? (Y * (Y + 1) / 2 + X) : (X * (X + 1) / 2 + Y) )
////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
void init_fake_data(){
  // cluster 1 of 5 points near (0, ..., 0) with std 1.0
  std::normal_distribution<float> distribution(0.0, 1.0);
  std::default_random_engine generator;
  for (int i = 0; i < N / 3; i++) {
    for (int j = 0; j < DIMENSIONS; j++) {
      data[i * DIMENSIONS + j] = distribution(generator);
    }
  }

  // cluster 2 of 5 points near (1000, ..., 1000) with std 0.01
  for (int i = N / 3; i < (2 * N) / 3; i++) {
    for (int j = 0; j < DIMENSIONS; j++) {
      data[i * DIMENSIONS + j] = distribution(generator)/100 + 1000;
    }
  }

  // cluster 3 of 6 points near (20000, ..., 20000) with std 500
  for (int i = (2 * N) / 3; i < N; i++) {
    for (int j = 0; j < DIMENSIONS; j++) {
      data[i * DIMENSIONS + j] = distribution(generator)*500 + 20000;
    }
  }
}

void sample_initial_solution(double *solution) {
  std::normal_distribution<float> distribution(0.0, 1.0);
  std::default_random_engine generator;

  for(int i = 0; i < N * DIMENSIONS_LOWER; i++) {
    solution[i] = distribution(generator) / 10000;
  }
}

double sum_arr_from_device(double* device_arr, int size) {
  double* host_arr = (double *)malloc(size * sizeof(double));
  checkCudaErrors(hipMemcpy(host_arr, device_arr, size * sizeof(double),
                             hipMemcpyDeviceToHost));
  double sum = 0;
  for(int i = 0; i < size; i++) {
    sum += host_arr[i];
  }
  free(host_arr);
  return sum;
}

void set_lerning_rates_device(double* d_lerning_rates, double initial_rate, int size) {
  double* host_lerning_rates = (double *)malloc(size * sizeof(double));
  for(int i = 0; i < size; i++) {
    host_lerning_rates[i] = initial_rate;
  }
  checkCudaErrors(hipMemcpy(d_lerning_rates, host_lerning_rates, size * sizeof(double),
                             hipMemcpyHostToDevice));
  free(host_lerning_rates);
}

int main(int argc, char **argv) {
  double *dData;
  double *distances_device;
  double *sigmas_device;
  double *sigmas_host;
  double *denominators_device; // for calculating pji - we can calculate it in the same kernel as sigmas
  double *p_asym_device; // p_i|j
  double *p_sym_device;  // p_ij
  double *p_asym_host;
  double *p_sym_host;

  double* solution;
  checkCudaErrors(hipMalloc(&dData, N * DIMENSIONS * sizeof(double)));

  checkCudaErrors(hipMalloc(&distances_device, N * (N + 1) / 2 * sizeof(double)));
  checkCudaErrors(hipMemset(distances_device, -1, N * (N + 1) / 2 * sizeof(double)));

  checkCudaErrors(hipMalloc(&denominators_device, N * sizeof(double)));
  checkCudaErrors(hipMemset(denominators_device, -1, N * sizeof(double)));

  checkCudaErrors(hipMalloc(&sigmas_device, N * sizeof(double)));
  checkCudaErrors(hipMemset(sigmas_device, -1, N * sizeof(double)));

  checkCudaErrors(hipMalloc(&p_asym_device, N * N * sizeof(double)));
  checkCudaErrors(hipMemset(p_asym_device, 0, N * N * sizeof(double)));

  checkCudaErrors(hipMalloc(&p_sym_device, N * (N + 1) / 2 * sizeof(double)));
  checkCudaErrors(hipMemset(p_sym_device, 0, N * (N + 1) / 2 * sizeof(double)));

  solution = (double *)malloc(N * DIMENSIONS_LOWER * sizeof(double));
  p_sym_host = (double *)malloc(N * (N + 1) / 2 * sizeof(double));
  p_asym_host = (double *)malloc(N * N * sizeof(double));
  sigmas_host = (double *)malloc(N * sizeof(double));
  // ^ triangle matrix of distances between points
  // 
  //   0 1 2 3 4
  // 0 0
  // 1 1 2
  // 2 3 4 5
  // 3 6 7 8 9
  // 4 10 11 12 13 14
  

  // init data and copy to device
  init_fake_data();
  sample_initial_solution(solution);
  checkCudaErrors(hipMemcpy(dData, data, N * DIMENSIONS * sizeof(double),
                             hipMemcpyHostToDevice));
                            
  // make timer
  StopWatchInterface *timer = NULL;
  sdkCreateTimer(&timer);

  // calculate distances
  sdkStartTimer(&timer);
  calculate_distances<<<(N + 1) / 2, THREADS>>>(dData, distances_device, DIMENSIONS, N);  
  checkCudaErrors(hipDeviceSynchronize());
  sdkStopTimer(&timer);
  std::cout << "Kernel time: " << sdkGetTimerValue(&timer) << std::endl;

  // debug Distance
  double *distances_host = (double *)malloc(N * (N + 1) / 2 * sizeof(double));
  checkCudaErrors(hipMemcpy(distances_host, distances_device, N * (N + 1) / 2 * sizeof(double),
                             hipMemcpyDeviceToHost));

  // calculating sigmas
  double perplexity = 5;
  double tolerance = 0.1;
  sdkCreateTimer(&timer);
  sdkStartTimer(&timer);
  calculate_sigmas<<<N, THREADS>>>(distances_device, sigmas_device, perplexity, tolerance, denominators_device, N);
  checkCudaErrors(hipDeviceSynchronize());
  sdkStopTimer(&timer);
  std::cout << "Kernel sigma time: " << sdkGetTimerValue(&timer) << std::endl;

  // calculating p_asym
  sdkCreateTimer(&timer);
  sdkStartTimer(&timer);
  calculate_p_asym<<<N, THREADS>>>(distances_device, sigmas_device, denominators_device, p_asym_device, N);
  checkCudaErrors(hipDeviceSynchronize());
  sdkStopTimer(&timer);
  std::cout << "Kernel p_asym time: " << sdkGetTimerValue(&timer) << std::endl;

  // now we can free distances_device and denominators_device
  checkCudaErrors(hipFree(distances_device));
  checkCudaErrors(hipFree(denominators_device));


  // calculating p_sym
  sdkCreateTimer(&timer);
  sdkStartTimer(&timer);
  calculate_p_sym<<<(N + 1) / 2, THREADS>>>(p_asym_device, p_sym_device, N);
  checkCudaErrors(hipDeviceSynchronize());
  sdkStopTimer(&timer);
  std::cout << "Kernel p_sym time: " << sdkGetTimerValue(&timer) << std::endl;
  checkCudaErrors(hipMemcpy(p_sym_host, p_sym_device, N * (N + 1) / 2 * sizeof(double),
                             hipMemcpyDeviceToHost)); // 


  // grtadient descent
  double* d_processed_distances;   // divided by q denoinator gives q (low dim affinites)
  double* d_solution;              // low dim solution
  double* d_solution_old;          // this will become handy for momentum
  double* d_denominator_for_block; // for calculating q
  double* d_grad;                  // gradient
  double* d_lerning_rates;         // learning rates for each parameter
  double* d_delta_bar;             // exponential average of partial derivatives
  double* d_kullback_leibler;      // for calculating kullback leibler divergence - just for curiosity
  checkCudaErrors(hipMalloc(&d_processed_distances, N * (N + 1) / 2 * sizeof(double)));
  checkCudaErrors(hipMalloc(&d_solution, N * DIMENSIONS_LOWER * sizeof(double)));
  checkCudaErrors(hipMalloc(&d_solution_old, N * DIMENSIONS_LOWER * sizeof(double)));
  checkCudaErrors(hipMalloc(&d_denominator_for_block, ((N + 1) / 2) * sizeof(double)));
  checkCudaErrors(hipMalloc(&d_grad, N * DIMENSIONS_LOWER * sizeof(double)));
  checkCudaErrors(hipMalloc(&d_lerning_rates, N * DIMENSIONS_LOWER * sizeof(double)));
  checkCudaErrors(hipMalloc(&d_delta_bar, N * DIMENSIONS_LOWER * sizeof(double)));
  checkCudaErrors(hipMalloc(&d_kullback_leibler, (N + 1) / 2 * sizeof(double)));

  checkCudaErrors(hipMemset(d_processed_distances, 0, N * (N + 1) / 2 * sizeof(double)));
  checkCudaErrors(hipMemset(d_solution, 3, N * DIMENSIONS_LOWER * sizeof(double)));

  checkCudaErrors(hipMemcpy(d_solution, solution, N * DIMENSIONS_LOWER * sizeof(double),
                             hipMemcpyHostToDevice));

  set_lerning_rates_device(d_lerning_rates, 100.0, N * DIMENSIONS_LOWER);
  // print initial (random) solution to check if it was copied correctly
  // for(int i = 0; i < N ; i++) {
  //   for(int j = 0; j < DIMENSIONS_LOWER; j++) {
  //     std::cout << "solution[" << i << "][" << j << "] = " << solution[i * DIMENSIONS_LOWER + j] << std::endl;
  //   }
  // }

  // parameters for gradient descent
  double alpha = 0.9; // momentum

  // for delta bar delta
  double kappa = 3.75;
  double fi = 0.1;
  double theta = 0.7;

  for(int i = 0; i < 100; i++) {
    calculate_distances<<<(N + 1) / 2, THREADS>>>(d_solution, d_processed_distances, DIMENSIONS_LOWER, N);
    checkCudaErrors(hipDeviceSynchronize());

    process_distances<<<(N + 1) / 2, THREADS>>>(d_processed_distances, d_denominator_for_block, N);
    checkCudaErrors(hipDeviceSynchronize());

    double denominator = 2 * sum_arr_from_device(d_denominator_for_block, (N + 1) / 2); // its important to
    // multiply by 2 because we are operating on half of the matrix, we want our array to sum up to 0.5 so whole matrix sums up to 1
    // just like in p_ij

    calculate_gradient<<<N, THREADS>>>(p_sym_device, d_processed_distances, d_solution, denominator, d_grad, N);

    // just for curiosity - calculate kulback leibler divergence
    __global__ void calculate_Kullback_Leibler(double *p, double *processed_distances, double denominator, double* partial_ans, int n);
    calculate_Kullback_Leibler<<<(N + 1) / 2, THREADS>>>(p_sym_device, d_processed_distances, denominator,d_kullback_leibler,  N);
    checkCudaErrors(hipDeviceSynchronize());
    double kullback_leibler = 2 * sum_arr_from_device(d_kullback_leibler, (N + 1) / 2); // cuz its half of the matrix
    std::cout << "Kullback Leibler divergence: " << kullback_leibler << std::endl;
    std::cout << "iteration: " << i << std::endl;
    // std::cout << "q summed: " << sum_arr_from_device(d_processed_distances, N * (N + 1) / 2) / denominator << std::endl;
    // std::cout << "denominator: " << denominator << std::endl;
    // std::cout << "p summed: " << sum_arr_from_device(p_sym_device, N * (N + 1) / 2) << std::endl;

    // update solution
    make_step_and_update_learning_rate<<<(N + 255) / 256, 256>>>(d_solution, d_solution_old, d_grad, d_lerning_rates, alpha,
                                                    theta, d_delta_bar, kappa, fi, DIMENSIONS_LOWER, N);

  }

  // print gradient
  double* grad = (double *)malloc(N * DIMENSIONS_LOWER * sizeof(double));
  checkCudaErrors(hipMemcpy(grad, d_grad, N * DIMENSIONS_LOWER * sizeof(double),
                             hipMemcpyDeviceToHost));

  // for(int i = 0; i < N ; i++) {
  //   for(int j = 0; j < DIMENSIONS_LOWER; j++) {
  //     std::cout << "grad[" << i << "][" << j << "] = " << grad[i * DIMENSIONS_LOWER + j] << std::endl;
  //   }
  // }

  checkCudaErrors(hipMemcpy(solution, d_solution, N * DIMENSIONS_LOWER * sizeof(double),
                             hipMemcpyDeviceToHost));
  // open file to save solution
  FILE *f = fopen("solution.txt", "w");
  fprintf(f, "x, y\n");
  for(int i = 0; i < N; i++) {
    for(int j = 0; j < DIMENSIONS_LOWER; j++) {
      fprintf(f, "%f", solution[i * DIMENSIONS_LOWER + j]);
      if(j != DIMENSIONS_LOWER - 1)
        fprintf(f, ", ");
    }
    fprintf(f, "\n");
  }
  // free memory
  checkCudaErrors(hipFree(dData));
  // checkCudaErrors(hipFree(distances_device));
  checkCudaErrors(hipFree(sigmas_device));
  checkCudaErrors(hipFree(p_asym_device));
  checkCudaErrors(hipFree(p_sym_device));
  free(p_sym_host);
  free(p_asym_host);
  free(sigmas_host);
  free(solution);
  
  sdkDeleteTimer(&timer);
  // finish
  exit(0);
}
