#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

// includes
#include <hip/hip_runtime_api.h>  // helper functions for CUDA error checking and initialization
#include <helper_functions.h>  // helper for shared functions common to CUDA Samples

#include <hip/hip_runtime.h>

#include <cassert>
#include <iostream>
#include <memory>
#include <stdio.h>

#include "helper_functions.h"
#include "consts.h"
// this macro is linear regarding smaller number and quadratic regarding the bigger number
#define TRIANGLE(X, Y) ( X < Y ? ( (Y) * ( (Y)  + 1) / 2 + (X) ) : ( (X) * ( (X) + 1) / 2 + (Y) ) ) 
// they must be in prenthesis because if X = var1 + var2 * var3, then the macro would expand to X < var1 + var2 * var3 < Y

//   0 1 2 3 4
// 0 0
// 1 1 2
// 2 3 4 5
// 3 6 7 8 9
// 4 10 11 12 13 14
// 5 15 16 17 18 19 20

__device__ float l2_dist_sq(float *a, float *b, int n) {
  float sum = 0;
  int index;
  for (int i = 0; i < n; i++) {
    index = i;
    sum += (a[index] - b[index]) * (a[index] - b[index]); // TODO try to diverge memory access like a[(i+c)%n] edit: tried and no difference
  }
  return sum;
}



// each block is responsible a row x and N - x so each block calculates N + 1 distances
__global__ void calculate_distances(float *d_data,float* distances, int dim, int n) {
  // int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x;
  // calculate distance to each data point with lower index

  int triangle_index = 0;

  // row myPoint
  int myPoint = blockIdx.x;
  int otherPoint = myPoint - 1 - threadIdx.x; // this version with divergent memory access is 3 x faster (wow)
  while(otherPoint >= 0 && myPoint < n){
    triangle_index = TRIANGLE(myPoint, otherPoint);
    distances[triangle_index] = l2_dist_sq(d_data + myPoint * dim, d_data + otherPoint * dim, dim);
    otherPoint -= stride;
  }

  // row N - myPoint
  myPoint = n - myPoint - 1;
  otherPoint = myPoint - 1 - threadIdx.x;
  while(otherPoint >= 0 && myPoint < n){
    triangle_index = TRIANGLE(myPoint, otherPoint);
    distances[triangle_index] = l2_dist_sq(d_data + myPoint * dim, d_data + otherPoint * dim, dim);
    otherPoint -= stride;
  }

}

__global__ void calculate_distances_tiled(float *d_data,float* distances, int dim, int n){
  extern __shared__ float s[];
  float *chunk_x = s;
  float *chunk_y = s + dim * TILE_WIDTH;

  int bx = blockIdx.x;
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  int stride = blockDim.x * blockDim.y;
  int in_block_linear_index = tx + ty * blockDim.x;

  // not all blocks are needed
  if(bx > by){
    return; 
  }

  // colaborative loading

  // first chunk x
  for(int i = in_block_linear_index; i < dim * TILE_WIDTH; i += stride){
    chunk_x[i] = d_data[(bx * blockDim.x * dim) + i];
  }
  // second chunk y
  for(int i = in_block_linear_index; i < dim * TILE_WIDTH; i += stride){
    chunk_y[i] = d_data[(by * blockDim.y * dim) + i];
  }

  __syncthreads();

  // thread divergence saddly
  if(bx == by && tx >= ty){
    return;
  }

  // calculate distance
  float distance = l2_dist_sq(chunk_x + tx * dim, chunk_y + ty * dim, dim);
  int triangle_index = TRIANGLE(bx * blockDim.x + tx, by * blockDim.y + ty);

  // write to global memory
  distances[triangle_index] = distance;
}

// each block is responsible for a single value of sigma for p_{j|blockId} for all j
__global__ void calculate_sigmas(float *distances_sq, float *sigmas, float perp, float tolerance, float* dominators, int n) 
{
  int blockId = blockIdx.x;
  int stride = blockDim.x;
  __shared__ float sigma;
  __shared__ float shared_denominator[THREADS];
  __shared__ float sum_of_numerators_logs[THREADS];
  __shared__ bool done;

  //only for the first thread in the block
  if(threadIdx.x == 0){
    done = false;
    sigma = 1; // initial guess
  }
  float sigma_upper_bound = sigma;
  float sigma_lower_bound = sigma;
  bool lower_bound_found = false;
  bool upper_bound_found = false;

  __syncthreads(); // BUG FIXED HERE
  while (!done)
  {  
    __syncthreads();
    float my_denominator = 0;
    float my_sum_of_numerators_logs = 0;
    int i = threadIdx.x;
    float temp = 0;
    float temp_exp = 0;
    while(i < n){
      if(i != blockId){
        int index = TRIANGLE(blockId, i);
        temp = -(distances_sq[index] / (2 * sigma * sigma));
        temp_exp = exp(temp);
        // compiler would probably optimize this to a single exp call 
        my_denominator += temp_exp;
        my_sum_of_numerators_logs += temp_exp * temp; 
      }
      i += stride;
    }

    shared_denominator[threadIdx.x] = my_denominator;
    sum_of_numerators_logs[threadIdx.x] = my_sum_of_numerators_logs;

    int limit = THREADS / 2;
    __syncthreads();

    while ( limit > 0)
    {
      if(threadIdx.x < limit){
        shared_denominator[threadIdx.x] += shared_denominator[threadIdx.x + limit];
        sum_of_numerators_logs[threadIdx.x] += sum_of_numerators_logs[threadIdx.x + limit];
        // assert(shared_denominator[threadIdx.x] >= 0);
      }
      limit /= 2;
      __syncthreads();
    }

    __syncthreads();

    if(threadIdx.x == 0){
      
      float shannon_entropy;
      if(shared_denominator[0] != 0){
       shannon_entropy = - (sum_of_numerators_logs[0] / shared_denominator[0]) + log(shared_denominator[0]);
      } else {
        shannon_entropy = 0;
      }

      float perplexity = exp(shannon_entropy * log(2.0));
      float diff = perplexity - perp;
      float diff_abs = diff > 0 ? diff : -diff;

      if(diff_abs < tolerance ){
        sigmas[blockId] = sigma;
        dominators[blockId] = shared_denominator[0];
        done = true; // this will break the while loop
      } else {
        if(lower_bound_found && upper_bound_found){
          if(diff > 0){
            sigma_upper_bound = sigma;
          } else {
            sigma_lower_bound = sigma;
          }
          sigma = (sigma_upper_bound + sigma_lower_bound) / 2;
        }
        else if(diff > 0){
          sigma_upper_bound = sigma;
          upper_bound_found = true;
          sigma /= 2;
        } else {
          sigma_lower_bound = sigma;
          lower_bound_found = true;
          sigma *= 2;
        }

      }
      
    }

  }
}

__global__ void calculate_p_asym(float *distances, float *sigmas, float *denominators, float *p_asym, int n){
  int i = blockIdx.x;
  int stride = blockDim.x;

  float denominator = denominators[i];
  float sigma = sigmas[i];
  int j = threadIdx.x;


  while (j < n){
    if(i != j){
      int triangle_index = TRIANGLE(i, j);
      p_asym[i * n + j] = exp(-distances[triangle_index] / (2 * sigma * sigma)) / denominator;
    }
    j += stride;
  }
  
}

__global__ void calculate_p_sym(float *p_asym, float *p_sym, int n){
  // int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x;
  // calculate distance to each data point with lower index

  int triangle_index = 0;

  // row myPoint
  int i = blockIdx.x;
  int j = i - 1 - threadIdx.x; // this version with divergent memory access is 3 x faster (wow)
  while(j >= 0 && i < n){
    triangle_index = TRIANGLE(i, j);
    float p = ((p_asym[i * n + j] + p_asym[j * n + i]) / (2 * n));
    p *= P_MULTIPLIER;
    
    p_sym[triangle_index] = p;
    j -= stride;
  }

  // row N - myPoint
  i = n - i - 1;
  j = i - 1 - threadIdx.x;
  // yes this is the same code as above but I don't want to make a function for this
  while(j >= 0 && i < n){
    triangle_index = TRIANGLE(i, j);
    float p = ((p_asym[i * n + j] + p_asym[j * n + i]) / (2 * n));
    p *= P_MULTIPLIER;

    p_sym[triangle_index] = p;
    j -= stride;
  }
}

// about 2 times slower than calculate_distances for dim=2
__global__ void process_distances(float *distances, float*denominator_for_block, int n){
  int stride = blockDim.x;
  int triangle_index = 0;
  __shared__ float shared_denominator[THREADS];
  shared_denominator[threadIdx.x] = 0;

  // row i
  int i = blockIdx.x;
  int j = i - 1 - threadIdx.x; // this version with divergent memory access is 3 x faster (wow)
  while(j >= 0 && i < n){
    triangle_index = TRIANGLE(i, j);
    
    distances[triangle_index] = 1 / (1 + distances[triangle_index]); // TODO try to store result in different array
    shared_denominator[threadIdx.x] += distances[triangle_index];

    j -= stride;
  }


  // row N - i
  i = n - i - 1;
  j = i - 1 - threadIdx.x;
  // yes this is the same code as above but I don't want to make a function for this
  while(j >= 0 && i < n){
    triangle_index = TRIANGLE(i, j);
    
    distances[triangle_index] = 1 / (1 + distances[triangle_index]);
    shared_denominator[threadIdx.x] += distances[triangle_index];
    
    j -= stride;
  }
  

  int limit = THREADS / 2;
   __syncthreads();

    while ( limit > 0)
    {
      if(threadIdx.x < limit){
        shared_denominator[threadIdx.x] += shared_denominator[threadIdx.x + limit];
      }
      limit /= 2;
      __syncthreads();
    }

    __syncthreads();
  
  if(threadIdx.x == 0){
    denominator_for_block[blockIdx.x] = shared_denominator[0];
  }

}

__global__ void calculate_and_process_distances(float *d_data,float* distances, float*denominator_for_block, int dim, int n){
  // int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x;
  int triangle_index = 0;

  __shared__ float shared_denominator[THREADS];
  shared_denominator[threadIdx.x] = 0;

  // Calculate distances and take 1 / (1 + distance) for each distance

  // row myPoint
  int myPoint = blockIdx.x;
  int otherPoint = myPoint - 1 - threadIdx.x; // this version with divergent memory access is 3 x faster (wow)
  while(otherPoint >= 0 && myPoint < n){
    triangle_index = TRIANGLE(myPoint, otherPoint);
    float result = 1 /(1 + l2_dist_sq(d_data + myPoint * dim, d_data + otherPoint * dim, dim));
    distances[triangle_index] =  result;
    shared_denominator[threadIdx.x] += result; // for calculating denominator
    otherPoint -= stride;
  }
  // row N - myPoint
  myPoint = n - myPoint - 1;
  otherPoint = myPoint - 1 - threadIdx.x;
  while(otherPoint >= 0 && myPoint < n){
    triangle_index = TRIANGLE(myPoint, otherPoint);
    float result = 1 / (1 + l2_dist_sq(d_data + myPoint * dim, d_data + otherPoint * dim, dim));
    distances[triangle_index] = result;
    shared_denominator[threadIdx.x] += result; // for calculating denominator
    otherPoint -= stride;
  }

  // calculate denominator

  int limit = THREADS / 2;
   __syncthreads();

    while ( limit > 0)
    {
      if(threadIdx.x < limit){
        shared_denominator[threadIdx.x] += shared_denominator[threadIdx.x + limit];
      }
      limit /= 2;
      __syncthreads();
    }

    __syncthreads();
  
  if(threadIdx.x == 0){
    denominator_for_block[blockIdx.x] = shared_denominator[0];
  }
}


__global__ void calculate_gradient(float *p, float *processed_distances, float *y, float denominator, float *grad, int n){
  int i = blockIdx.x;
  int stride = blockDim.x;

  __shared__ float shared_grad[THREADS * DIMENSIONS_LOWER];

  for(int i = 0; i < DIMENSIONS_LOWER; i++){
    shared_grad[threadIdx.x * DIMENSIONS_LOWER + i] = 0;
  }

  // each thread is zeroing out memory it will use so there is no need for synchronization 
  int j = threadIdx.x;
  
  while(j < n){
    if(i != j){
      int triangle_index = TRIANGLE(i, j);
      float q = processed_distances[triangle_index] / denominator;
      // TODO use normal square array
      for(int k = 0; k < DIMENSIONS_LOWER; k++){
        shared_grad[threadIdx.x * DIMENSIONS_LOWER + k] += 4 * (p[triangle_index] - q) 
                * (y[i * DIMENSIONS_LOWER + k] - y[j * DIMENSIONS_LOWER + k]) *  processed_distances[triangle_index];
      }
    }
    j += stride;
  }

  int limit = THREADS / 2;
   __syncthreads();

    while ( limit > 0)
    {
      if(threadIdx.x < limit){
        for(int i = 0; i < DIMENSIONS_LOWER; i++){
           shared_grad[threadIdx.x * DIMENSIONS_LOWER + i] += shared_grad[(threadIdx.x + limit) * DIMENSIONS_LOWER + i];
        }
      }
      limit /= 2;
      __syncthreads();
    }

    __syncthreads();

    if(threadIdx.x == 0){
      for(int i = 0; i < DIMENSIONS_LOWER; i++){
        grad[blockIdx.x * DIMENSIONS_LOWER + i] = shared_grad[i];
      }
    }

}

__global__ void calculate_Kullback_Leibler(float *p, float *processed_distances, float denominator, float* partial_ans, int n){
  int stride = blockDim.x;
  int triangle_index = 0;
  __shared__ float shared_kullback[THREADS];
  shared_kullback[threadIdx.x] = 0;

  // row i
  int i = blockIdx.x;
  int j = i - 1 - threadIdx.x; 
  while(j >= 0 && i < n){
    triangle_index = TRIANGLE(i, j);
    
    float q = processed_distances[triangle_index] / denominator;
    if(p[triangle_index] != 0)
      shared_kullback[threadIdx.x] += p[triangle_index] * log(p[triangle_index] / q);

    j -= stride;
  }

  // row N - i
  i = n - i - 1;
  j = i - 1 - threadIdx.x;
  // yes this is the same code as above but I don't want to make a function for this
  while(j >= 0 && i < n){
    triangle_index = TRIANGLE(i, j);
    
    float q = processed_distances[triangle_index] / denominator;
    if(p[triangle_index] != 0)
      shared_kullback[threadIdx.x] += p[triangle_index] * log(p[triangle_index] / q);
    
    j -= stride;
  }
  

  int limit = THREADS / 2;
   __syncthreads();

    while ( limit > 0)
    {
      if(threadIdx.x < limit){
        shared_kullback[threadIdx.x] += shared_kullback[threadIdx.x + limit];
      }
      limit /= 2;
      __syncthreads();
    }

    __syncthreads();
  
  if(threadIdx.x == 0){
    partial_ans[blockIdx.x] = shared_kullback[0];
  }

}

__global__ void make_step_and_update_learning_rate(float *y, float *old_y, float *grad, float *learning_rates, float alpha,
                                                    float theta, float *d_delta_bar,float kappa, float fi, int dim_lower, int n)
{
    int j = threadIdx.x + blockIdx.x * blockDim.x;

  if(j < n){
    for(int k = 0; k < dim_lower; k++){
      // index of the element in the y array
      int index = j * dim_lower + k;

      // update y
      float momentum = alpha * (y[index] - old_y[index]);
      old_y[index] = y[index]; // update old_y
      y[index] = y[index] - learning_rates[j] * grad[index] + momentum; // TODO add noise

      // update learning rate
      if(grad[index] * d_delta_bar[index] > 0){
        learning_rates[index] = learning_rates[index] + kappa < MAX_LEARNING_RATE ? learning_rates[index] + kappa : MAX_LEARNING_RATE;
      }
      else{
        learning_rates[index] = learning_rates[index] * fi; 
      }

      // update average gradient
      d_delta_bar[index] = (1 - theta) * grad[index] + theta * d_delta_bar[index];
    }
  }
}