#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

// includes
#include <hip/hip_runtime_api.h>  // helper functions for CUDA error checking and initialization
#include <helper_functions.h>  // helper for shared functions common to CUDA Samples

#include <hip/hip_runtime.h>

#include <cassert>
#include <iostream>
#include <memory>
#include <stdio.h>

#include "helper_functions.h"
#include "consts.h"
#define TRIANGLE(X, Y) ( X < Y ? (Y * (Y + 1) / 2 + X) : (X * (X + 1) / 2 + Y) )
//   0 1 2 3 4
// 0 0
// 1 1 2
// 2 3 4 5
// 3 6 7 8 9

__device__ double l2_dist_sq(double *a, double *b, int n) {
  double sum = 0;
  int index;
  for (int i = 0; i < n; i++) {
    index = i;
    sum += (a[index] - b[index]) * (a[index] - b[index]); // TODO try to diverge memory access like a[(i+c)%n] edit: tried and no difference
  }
  return sum;
}


// each block is responsible a column x and N - x so each block calculates N + 1 distances
__global__ void calculate_distances(double *d_data,double* distances, int n) {
  // int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x;
  // calculate distance to each data point with lower index

  int triangle_index = 0;

  // column myPoint
  int myPoint = blockIdx.x;
  int otherPoint = myPoint - 1 - threadIdx.x; // this version with divergent memory access is 3 x faster (wow)
  while(otherPoint >= 0 && myPoint < n){
    triangle_index = TRIANGLE(myPoint, otherPoint);
    distances[triangle_index] = l2_dist_sq(d_data + myPoint * DIMENSIONS, d_data + otherPoint * DIMENSIONS, DIMENSIONS);
    otherPoint -= stride;
  }

  // column N - myPoint
  myPoint = n - myPoint - 1;
  otherPoint = myPoint - 1 - threadIdx.x;
  while(otherPoint >= 0 && myPoint < n){
    triangle_index = TRIANGLE(myPoint, otherPoint);
    distances[triangle_index] = l2_dist_sq(d_data + myPoint * DIMENSIONS, d_data + otherPoint * DIMENSIONS, DIMENSIONS);
    otherPoint -= stride;
  }

}

// each block is responsible for a single value of sigma for p_{j|blockId} for all j
__global__ void calculate_sigmas(double *distances_sq, double *sigmas, double perp, double tolerance, int n) 
{
  int blockId = blockIdx.x;
  int stride = blockDim.x;
  __shared__ double sigma;
  __shared__ double shared_denominator[THREADS];
  __shared__ double sum_of_numerators_logs[THREADS];
  __shared__ bool done;

  //only for the first thread in the block
  if(threadIdx.x == 0){
    done = false;
    sigma = 1; // initial guess
  }
  double sigma_upper_bound = sigma;
  double sigma_lower_bound = sigma;
  bool lower_bound_found = false;
  bool upper_bound_found = false;

  __syncthreads(); // BUG FIXED HERE
  while (!done)
  {  
    __syncthreads();
    double my_denominator = 0;
    double my_sum_of_numerators_logs = 0;
    int i = threadIdx.x;
    double temp = 0;
    double temp_exp = 0;
    while(i < n){
      if(i != blockId){
        int index = TRIANGLE(blockId, i);
        temp = -(distances_sq[index] / (2 * sigma * sigma));
        temp_exp = exp(temp);
        // compiler would probably optimize this to a single exp call 
        my_denominator += temp_exp;
        my_sum_of_numerators_logs += temp_exp * temp; 
      }
      i += stride;
    }

    shared_denominator[threadIdx.x] = my_denominator;
    sum_of_numerators_logs[threadIdx.x] = my_sum_of_numerators_logs;

    int limit = THREADS / 2;
    __syncthreads();

    while ( limit > 0)
    {
      if(threadIdx.x < limit){
        shared_denominator[threadIdx.x] += shared_denominator[threadIdx.x + limit];
        sum_of_numerators_logs[threadIdx.x] += sum_of_numerators_logs[threadIdx.x + limit];
        // assert(shared_denominator[threadIdx.x] >= 0);
      }
      limit /= 2;
      __syncthreads();
    }

    __syncthreads();

    if(threadIdx.x == 0){
      
      double shannon_entropy;
      if(shared_denominator[0] != 0){
       shannon_entropy = - (sum_of_numerators_logs[0] / shared_denominator[0]) + log(shared_denominator[0]);
      } else {
        shannon_entropy = 0;
      }

      double perplexity = exp(shannon_entropy * log(2.0));
      double diff = perplexity - perp;
      double diff_abs = diff > 0 ? diff : -diff;

      if(diff_abs < tolerance ){
        sigmas[blockId] = sigma;
        done = true; // this will break the while loop
      } else {
        if(lower_bound_found && upper_bound_found){
          if(diff > 0){
            sigma_upper_bound = sigma;
          } else {
            sigma_lower_bound = sigma;
          }
          sigma = (sigma_upper_bound + sigma_lower_bound) / 2;
        }
        else if(diff > 0){
          sigma_upper_bound = sigma;
          upper_bound_found = true;
          sigma /= 2;
        } else {
          sigma_lower_bound = sigma;
          lower_bound_found = true;
          sigma *= 2;
        }

      }
      
    }

  }
}